#include <iostream>
#include "host.hpp"
#include "kernel.cuh"

#define MATRIX_M 1024
#define MATRIX_N 1024
#define MATRIX_K 1024

#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
    if (stat != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
    }
}

int main() {
    float *A;
    float *B;
    float *C;

    cudaErrCheck(hipMalloc((void **) &a_fp32, MATRIX_M * MATRIX_K * sizeof(float)));
    A = (float *) malloc(MATRIX_M * MATRIX_K * sizeof(float));
    B = (float *) malloc(MATRIX_K * MATRIX_N * sizeof(float));
    B = (float *) malloc(MATRIX_K * MATRIX_N * sizeof(float));

#pragma omp parallel for
    for (int i = 0; i < 100; ++i) {
        std::cout << i << std::endl;
    }

    return 0;
}

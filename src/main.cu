#include <cstdio>

#include "CudaTimeCalculator.hpp"
#include "kernel.cuh"
#include "cudaErrorCheck.cuh"

#define VEC_SIZE 1048576

int main() {
    float *vecA;
    float *vecB;
    float *vecC;

    cudaErrCheck(hipMalloc((void **) &vecA, VEC_SIZE * sizeof(float)));
    cudaErrCheck(hipMalloc((void **) &vecB, VEC_SIZE * sizeof(float)));
    cudaErrCheck(hipMalloc((void **) &vecC, VEC_SIZE * sizeof(float)));

    CudaTimeCalculator timeCalculator;

    const int numThreadPerBlocks = 1024;
    const int numBlocks = (VEC_SIZE + numThreadPerBlocks - 1) / numThreadPerBlocks;

    timeCalculator.startClock();
    vecAdd<float><<<numBlocks, numThreadPerBlocks>>>(VEC_SIZE, vecA, vecB, vecC);
    timeCalculator.endClock();

    printf("VEC_SIZE : %d\n"
           "vecAdd Function time : %fms\n", VEC_SIZE, timeCalculator.getTime());

    cudaErrCheck(hipFree(vecA));
    cudaErrCheck(hipFree(vecB));
    cudaErrCheck(hipFree(vecC));

    return 0;
}

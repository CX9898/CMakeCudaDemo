#include <iostream>
#include "host.hpp"
#include "kernel.cuh"
#include "hip/hip_runtime_api.h"

#define MATRIX_M 1024
#define MATRIX_N 1024
#define MATRIX_K 1024

int main() {
    float aHost;
    float bHost;
    float cHost;

    float *aDev;
    float *bDev;
    float *cDev;

    checkCudaErrors(hipMalloc((void **) &aDev, MATRIX_M * MATRIX_K * sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &bDev, MATRIX_K * MATRIX_N * sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &cDev, MATRIX_M * MATRIX_N * sizeof(float)));


    return 0;
}

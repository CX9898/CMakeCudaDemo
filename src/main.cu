#include <cstdio>

#include "kernel.cuh"
#include "cudaErrorCheck.cuh"

#define VEC_SIZE 1048576

int main() {
    float *vecA;
    float *vecB;
    float *vecC;

    cudaErrCheck(hipMalloc((void **) &vecA, VEC_SIZE * sizeof(float)));
    cudaErrCheck(hipMalloc((void **) &vecB, VEC_SIZE * sizeof(float)));
    cudaErrCheck(hipMalloc((void **) &vecC, VEC_SIZE * sizeof(float)));

    hipEvent_t star, stop;
    cudaErrCheck(hipEventCreate(&star));
    cudaErrCheck(hipEventCreate(&stop));

    const int numThreadPerBlocks = 1024;
    const int numBlocks = (VEC_SIZE + numThreadPerBlocks - 1) / numThreadPerBlocks;

    cudaErrCheck(hipEventRecord(star));
    vecAdd<float><<<numBlocks, numThreadPerBlocks>>>(VEC_SIZE, vecA, vecB, vecC);
    cudaErrCheck(hipEventRecord(stop));
    hipDeviceSynchronize();

    float time;
    cudaErrCheck(hipEventElapsedTime(&time, star, stop));

    printf("VEC_SIZE : %d\n"
           "vecAdd Function time : %fms\n", VEC_SIZE, time);

    cudaErrCheck(hipEventDestroy(star));
    cudaErrCheck(hipEventDestroy(stop));

    cudaErrCheck(hipFree(vecA));
    cudaErrCheck(hipFree(vecB));
    cudaErrCheck(hipFree(vecC));

    return 0;
}

#include <cstdio>

#include "kernel.cuh"
#include "hip/hip_runtime_api.h"

#define VEC_SIZE 1048576

int main() {
    float *a;
    float *b;
    float *c;

    checkCudaErrors(hipMalloc((void **) &a, VEC_SIZE * sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &b, VEC_SIZE * sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &c, VEC_SIZE * sizeof(float)));

    hipEvent_t star, stop;
    checkCudaErrors(hipEventCreate(&star));
    checkCudaErrors(hipEventCreate(&stop));

    const int numThreadPerBlocks = 1024;
    const int numBlocks = (VEC_SIZE + numThreadPerBlocks - 1) / numThreadPerBlocks;

    checkCudaErrors(hipEventRecord(star));
    vecAdd<float><<<numBlocks, numThreadPerBlocks>>>(VEC_SIZE, a, b, c);
    checkCudaErrors(hipEventRecord(stop));
    hipDeviceSynchronize();

    float time;
    checkCudaErrors(hipEventElapsedTime(&time, star, stop));

    printf("time : %fms\n", time);

    checkCudaErrors(hipEventDestroy(star));
    checkCudaErrors(hipEventDestroy(stop));

    hipFree(a);
    hipFree(b);
    hipFree(c);

    return 0;
}

#include <iostream>
#include "host.hpp"
#include "kernel.cuh"
#include "hip/hip_runtime_api.h"

#define MATRIX_M 1024
#define MATRIX_N 1024
#define MATRIX_K 1024

int main() {
    float *A;
    float *B;
    float *C;

    checkCudaErrors(hipMalloc((void **) &A, MATRIX_M * MATRIX_K * sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &B, MATRIX_K * MATRIX_N * sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &C, MATRIX_M * MATRIX_N * sizeof(float)));


    return 0;
}

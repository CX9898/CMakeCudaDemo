#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include <hip/hip_fp16.h>

template<typename T>
__global__ void vecAdd(const int n, const T *a, const T *b, T *c) {
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= n) {
        return;
    }

    c[tid] = a[tid] + b[tid];
}

template __global__ void vecAdd<float>(const int, const float*, const float*, float*);
template __global__ void vecAdd<double>(const int, const double*, const double*, double*);
template __global__ void vecAdd<half>(const int, const half*, const half*, half*);
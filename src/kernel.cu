#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>

#include "kernel.cuh"

template<typename T>
__global__ void vecAdd(const int n, const T *vecA, const T *vecB, T *result) {
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= n) {
        return;
    }

    result[tid] = vecA[tid] + vecB[tid];
}

template __global__ void vecAdd<half>(const int, const half *, const half *, half *);
template __global__ void vecAdd<float>(const int, const float *, const float *, float *);
template __global__ void vecAdd<double>(const int, const double *, const double *, double *);
template __global__ void vecAdd<int>(const int, const int *, const int *, int *);